
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  /*
   * デバイスを照会するには、まず、デバイス ID が必要です。
   */

  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  /*
   * `props` には、現在のデバイスに関するいくつかのプロパティが含まれています。
   */

  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;


  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}
