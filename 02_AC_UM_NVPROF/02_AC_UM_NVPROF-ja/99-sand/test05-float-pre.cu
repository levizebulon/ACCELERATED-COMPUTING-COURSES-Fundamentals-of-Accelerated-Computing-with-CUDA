#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void kernel(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = a[i] * 2.0f;
  }
}


void init(float *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 2.0f;
  }
}

int main(int argc, char **argv)
{
  int deviceId;
  hipGetDevice(&deviceId);

  int E = 20;
  if (argc > 1) E = atoi(argv[1]);
  
  int N = 2<<E;
  printf("N is 2<<%d: %d\n", E, 2<<E);

  float *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);
  init(a, N);

  hipMemPrefetchAsync(a, size, deviceId);

  size_t threadsPerBlock = 256;
  size_t numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  kernel<<<numberOfBlocks, threadsPerBlock>>>(a, N);
  hipDeviceSynchronize();
  printf("Done\n");
}
