
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  /*
   * 先ほどのコード (現在コメントアウト済み) では、
   * `a` の範囲外の要素にアクセスしようとしていました。
   */

  // for (int i = idx; i < N + stride; i += stride)
  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  /*
   * 先ほどのコード (現在コメントアウト済み) では、
   * ブロックあたりの最大数 (1024) を超えるスレッドで
   * カーネルを起動しようとしていました。
   */

  size_t threads_per_block = 1024;
  /* size_t threads_per_block = 2048; */
  size_t number_of_blocks = 32;

  hipError_t syncErr, asyncErr;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);

  /*
   * 上記のカーネルの起動エラーと、非同期の `doubleElements`
   * カーネルの実行中に発生したエラーの両方をキャッチします。
   */

  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();

  /*
   * エラーが存在する場合は、そのエラーを出力します。
   */

  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
