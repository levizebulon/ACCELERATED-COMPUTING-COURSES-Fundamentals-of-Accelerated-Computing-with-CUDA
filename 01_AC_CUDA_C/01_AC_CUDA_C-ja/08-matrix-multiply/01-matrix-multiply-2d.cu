
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  64

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  /*
   * このカーネルを構築します。
   */
}

/*
 * この CPU 関数は既に動作するようになっており、
 * 実行すると matrixMulGPU カーネルの構築作業を
 * 検証するためのソリューション マトリックスが作成されます。
 */

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu; // CPU, GPU 双方のソリューション マトリクス

  int size = N * N * sizeof (int); // N x N 行列のバイト数

  // メモリの確保
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // メモリの初期化
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  /*
   * 上の matrixMulGPU で使用できる 2D 値を
   * `threads_per_block` と `number_of_block` に代入します。
   */

  dim3 threads_per_block;
  dim3 number_of_blocks;

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize();

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}
